#include "hip/hip_runtime.h"
#include "gpu_fst_engine.h"
#include "shared.h"
typedef struct Return_message{
	uint32_t var_0;
	uint32_t var_1;
	uint32_t nextState;
} Return_message;

__global__
void dense_csr_function_group_global( uint32_t ** input_base, uint32_t ** input_length, uint32_t ** input,
						uint32_t **output_base, uint32_t ** output, uint32_t * test_output){
	uint32_t globalID = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t startStack_shared = threadIdx.x * 0;
	uint32_t base_0 = input_base[globalID][0];
	uint32_t length_0 = input_length[globalID][0];
	uint32_t processed_0 = 0;
	uint32_t base_1 = input_base[globalID][1];
	uint32_t length_1 = input_length[globalID][1];
	uint32_t processed_1 = 0;
	uint32_t currentState = 0;
	uint32_t var_0 =  0;
	uint32_t var_1 =  0;
	uint32_t var_2 =  0;
	uint32_t var_3 =  0;
	uint32_t var_4 =  0;
	uint32_t var_5 =  0;
	uint32_t printed = 0;
	bool done = false;
	uint32_t cycle =0;
	Return_message message;
#ifdef DEBUG
	while ( cycle < 20){
		printf("=============%d=============\n", cycle);
		printf("Current State : %d\n", currentState);
		printf("Var[0]: %d\n", var_0);
		printf("Var[1]: %d\n", var_1);
		printf("Var[2]: %d\n", var_2);
		printf("Var[3]: %d\n", var_3);
		printf("Var[4]: %d\n", var_4);
		printf("Var[5]: %d\n", var_5);
		printf("Processed Input 0: %d/%d\n", processed_0, length_0);
		printf("Current Input: ");
		for ( uint32_t i =0 ; i < 20; i++){
			printf("%d ", input[0][base_0+i]);
}
		printf("\n");
		printf("Processed Input 1: %d/%d\n", processed_1, length_1);
		printf("Current Input: ");
		for ( uint32_t i =0 ; i < 20; i++){
			printf("%d ", input[1][base_1+i]);
}
		printf("\n");
		printf("Current Output: ");
		for ( uint32_t i =0 ; i < printed; i++){
			printf("%d ", test_output[i]);
		}
		printf("\n");
		cycle++;
#else
	while (  (!done) && ( currentState != 9)){
#endif
		if(currentState == 0){
				var_0= 0;
				var_2=input[0][base_0 + processed_0];
			processed_0++;
				var_2=input[0][base_0 + processed_0];
			processed_0++;
				var_4=input[0][base_0 + processed_0];
			processed_0++;
				var_1= 0;
				var_3= 0;
			currentState = 1;
		}
		else if (currentState ==1){
				var_3=input[1][base_1 + processed_1];
			currentState = 2;
			processed_1++;
		}
		else if (currentState ==2){
			if((var_3!=0)){
				currentState = 3;
			}
			else if((var_3 == 0)){
				currentState = 4;
			}
		}
		else if (currentState ==3){
			var_5 = var_5 + 1;
			currentState = 4;
		}
		else if (currentState ==4){
			var_1 = var_1 + 1;
			currentState = 5;
		}
		else if (currentState ==5){
			if((var_1 == var_2)){
				currentState = 6;
			}
			else if((var_1!= var_2)){
				currentState = 1;
			}
		}
		else if (currentState ==6){
				var_1= 0;
			var_0 = var_0 + 1;
#ifdef DEBUG
			test_output[printed] = var_5;
			printed++;
#else 
			output[globalID][0] = var_5;
#endif
			currentState = 7;
		}
		else if (currentState ==7){
			if((var_5 == var_4)){
				currentState = 8;
			}
			else if((var_5!= var_4)){
				currentState = 1;
			}
		}
		else if (currentState ==8){
				var_0= 0;
				var_1= 0;
				var_3= 0;
			currentState = 1;
		}
		else {
			currentState = 9;
		}
		if ((processed_0 >= length_0)&& (processed_1 >= length_1))
			done = true;
	}
}
void FSTGPU::dense_csr_function_group_host(){
	printf("Begin dense_csr_function_group_host \n ");
	printf(" Number of block: %d \n", number_of_block);
	printf(" Number of thread per block: %d\n", number_of_thread);
	hipEvent_t start_execution, stop_execution;
	hipEventCreate( &start_execution);
	hipEventCreate( &stop_execution);
	hipEventRecord( start_execution, 0);
	dense_csr_function_group_global<<< number_of_block,number_of_thread>>> ( partition_input_base, partition_input_length, input,  partition_output_base,  simulated_output, printed_output);
	hipEventRecord( stop_execution,0);
	hipEventSynchronize(stop_execution);
	float execution_time_ms = 0;
	hipEventElapsedTime(&execution_time_ms, start_execution, stop_execution);
	printf("Execution  Finished\n");
	hipError_t err = hipGetLastError();
	if ( err != hipSuccess )
		printf("CUDA Error: %s\n", hipGetErrorString(err));
	fprintf(stderr,"Execution Time %f ms\n", execution_time_ms);
}
