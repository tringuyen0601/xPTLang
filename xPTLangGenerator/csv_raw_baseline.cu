#include "hip/hip_runtime.h"
#include "gpu_fst_engine.h"
#include "shared.h"
typedef struct Return_message{
	uint32_t var_0;
	uint32_t var_1;
	uint32_t nextState;
} Return_message;

__global__
void csv_raw_baseline_global( uint32_t ** input_base, uint32_t ** input_length, uint32_t ** input,
						uint32_t **output_base, uint32_t ** output, uint32_t * test_output){
	uint32_t globalID = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t startStack_shared = threadIdx.x * 1;
	uint32_t base_0 = input_base[globalID][0];
	uint32_t length_0 = input_length[globalID][0];
	uint32_t processed_0 = 0;
	uint32_t currentState = 0;
	uint32_t var_0 =  0;
	uint32_t var_1 =  0;
	uint32_t var_2 =  0;
	__shared__ uint32_t stack [1 * STACK_DEPTH * THREAD_PER_BLOCK];
	for (uint32_t i = 0; i < 1; i++){
		 stack[ (threadIdx.x *1 +i)* STACK_DEPTH ] = 0;
	}
	__syncthreads();
	uint32_t printed = 0;
	bool done = false;
	uint32_t cycle =0;
	Return_message message;
#ifdef DEBUG
	while ( cycle < 20){
		printf("=============%d=============\n", cycle);
		printf("Current State : %d\n", currentState);
		printf("Stack[ %d]:depth %d\n", startVar_shared + 0, var_0);
		for ( uint32_t v = 0; v < var_0; v++){
			printf("%d, ", stack[(startVar_shared +0) * STACK_DEPTH + v]);
		}
		printf("\n");
		printf("Var[1]: %d\n", var_1);
		printf("Var[2]: %d\n", var_2);
		printf("Processed Input 0: %d/%d\n", processed_0, length_0);
		printf("Current Input: ");
		for ( uint32_t i =0 ; i < 20; i++){
			printf("%d ", input[0][base_0+i]);
}
		printf("\n");
		printf("Current Output: ");
		for ( uint32_t i =0 ; i < printed; i++){
			printf("%d ", test_output[i]);
		}
		printf("\n");
		cycle++;
#else
	while (  (!done) && ( currentState != 22)){
#endif
		if(currentState == 0){
			if((var_1 == 0)){
				currentState = 1;
			}
			else if((var_1!=0)){
				currentState = 18;
			}
		}
		else if (currentState ==1){
				var_2=input[0][base_0 + processed_0];
			currentState = 2;
			processed_0++;
		}
		else if (currentState ==2){
			if((var_2!=59)){
				currentState = 3;
			}
			else if((var_2 == 59)){
				currentState = 4;
			}
		}
		else if (currentState ==3){
				var_2=input[0][base_0 + processed_0];
			currentState = 2;
			processed_0++;
		}
		else if (currentState ==4){
			var_1 = var_1 + 1;
			currentState = 5;
		}
		else if (currentState ==5){
				var_2=input[0][base_0 + processed_0];
			currentState = 6;
			processed_0++;
		}
		else if (currentState ==6){
			if((var_2!=59)){
				currentState = 7;
			}
			else if((var_2 == 59)){
				currentState = 8;
			}
		}
		else if (currentState ==7){
				var_2=input[0][base_0 + processed_0];
			currentState = 6;
			processed_0++;
		}
		else if (currentState ==8){
			var_1 = var_1 + 1;
			currentState = 9;
		}
		else if (currentState ==9){
				var_2=input[0][base_0 + processed_0];
			currentState = 10;
			processed_0++;
		}
		else if (currentState ==10){
			if((var_2 == 2)){
				currentState = 11;
			}
			else if((var_2!=2)){
				currentState = 13;
			}
		}
		else if (currentState ==11){
				for( uint32_t i = 0; i < var_0; i++){
#ifdef DEBUG
				test_output[printed] = stack[(threadIdx.x * 1 + 0) * STACK_DEPTH + i];
				printed++;
#else 
				output[globalID][0] = stack[(threadIdx.x * 1 + 0) * STACK_DEPTH + i];
#endif
				}
			currentState = 12;
		}
		else if (currentState ==12){
				var_0=1;
				stack[(threadIdx.x * 1 + 0) * STACK_DEPTH ]=0;
			currentState = 14;
		}
		else if (currentState ==13){
				var_0=1;
				stack[(threadIdx.x * 1 + 0) * STACK_DEPTH ]=0;
			currentState = 14;
		}
		else if (currentState ==14){
				var_2=input[0][base_0 + processed_0];
			currentState = 15;
			processed_0++;
		}
		else if (currentState ==15){
			if((var_2!=10)){
				currentState = 16;
			}
			else if((var_2 == 10)){
				currentState = 0;
			}
		}
		else if (currentState ==16){
				var_2=input[0][base_0 + processed_0];
			currentState = 17;
			processed_0++;
		}
		else if (currentState ==17){
			var_1 = var_1 & 0;
			currentState = 15;
		}
		else if (currentState ==18){
				var_2=input[0][base_0 + processed_0];
			currentState = 19;
			processed_0++;
		}
		else if (currentState ==19){
			if((var_2!=59)){
				currentState = 20;
			}
			else if((var_2 == 59)){
				currentState = 21;
			}
		}
		else if (currentState ==20){
				var_2=input[0][base_0 + processed_0];
			currentState = 19;
			processed_0++;
		}
		else if (currentState ==21){
			var_1 = var_1 + 1;
			currentState = 0;
		}
		else {
			currentState = 22;
		}
		if ((processed_0 >= length_0))
			done = true;
	}
}
void FSTGPU::csv_raw_baseline_host(){
	printf("Begin csv_raw_baseline_host \n ");
	printf(" Number of block: %d \n", number_of_block);
	printf(" Number of thread per block: %d\n", number_of_thread);
	hipEvent_t start_execution, stop_execution;
	hipEventCreate( &start_execution);
	hipEventCreate( &stop_execution);
	hipEventRecord( start_execution, 0);
	csv_raw_baseline_global<<< number_of_block,number_of_thread>>> ( partition_input_base, partition_input_length, input,  partition_output_base,  simulated_output, printed_output);
	hipEventRecord( stop_execution,0);
	hipEventSynchronize(stop_execution);
	float execution_time_ms = 0;
	hipEventElapsedTime(&execution_time_ms, start_execution, stop_execution);
	printf("Execution  Finished\n");
	hipError_t err = hipGetLastError();
	if ( err != hipSuccess )
		printf("CUDA Error: %s\n", hipGetErrorString(err));
	fprintf(stderr,"Execution Time %f ms\n", execution_time_ms);
}
