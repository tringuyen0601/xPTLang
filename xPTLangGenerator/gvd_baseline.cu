#include "hip/hip_runtime.h"
#include "gpu_fst_engine.h"
typedef struct Return_message{
	uint32_t var_0;
	uint32_t var_1;
	uint32_t nextState;
} Return_message;

__global__
void gvd_baseline_global( uint32_t ** input_base, uint32_t ** input_length, uint32_t ** input,
						uint32_t **output_base, uint32_t ** output, uint32_t * test_output){
	uint32_t globalID = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t startStack_shared = threadIdx.x * 0;
	uint32_t base_0 = input_base[globalID][0];
	uint32_t length_0 = input_length[globalID][0];
	uint32_t processed_0 = 0;
	uint32_t currentState = 0;
	uint32_t var_0 =  0;
	uint32_t var_1 =  0;
	uint32_t var_2 =  0;
	uint32_t var_3 =  0;
	uint32_t var_4 =  0;
	uint32_t var_5 =  0;
	uint32_t printed = 0;
	bool done = false;
	uint32_t cycle =0;
	Return_message message;
#ifdef DEBUG
	while ( cycle < 20){
		printf("=============%d=============\n", cycle);
		printf("Current State : %d\n", currentState);
		printf("Var[0]: %d\n", var_0);
		printf("Var[1]: %d\n", var_1);
		printf("Var[2]: %d\n", var_2);
		printf("Var[3]: %d\n", var_3);
		printf("Var[4]: %d\n", var_4);
		printf("Var[5]: %d\n", var_5);
		printf("Processed Input 0: %d/%d\n", processed_0, length_0);
		printf("Current Input: ");
		for ( uint32_t i =0 ; i < 20; i++){
			printf("%d ", input[0][base_0+i]);
}
		printf("\n");
		printf("Current Output: ");
		for ( uint32_t i =0 ; i < printed; i++){
			printf("%d ", test_output[i]);
		}
		printf("\n");
		cycle++;
#else
	while (  (!done) && ( currentState != 23)){
#endif
		if(currentState == 0){
			var_1 = var_1 & 0;
			currentState = 1;
		}
		else if (currentState ==1){
			var_2 = var_2 & 0;
			currentState = 2;
		}
		else if (currentState ==2){
			var_3 = var_3 & 0;
			currentState = 3;
		}
		else if (currentState ==3){
			var_4 = var_4 & 0;
			currentState = 4;
		}
		else if (currentState ==4){
			var_5 = var_5 & 0;
			currentState = 5;
		}
		else if (currentState ==5){
				var_0=input[0][base_0 + processed_0];
			currentState = 6;
			processed_0++;
		}
		else if (currentState ==6){
			var_2 = var_0 & 127;
			currentState = 7;
		}
		else if (currentState ==7){
			var_1 = var_0 & 128;
			currentState = 8;
		}
		else if (currentState ==8){
			var_5 = var_4 * 7;
			currentState = 9;
		}
		else if (currentState ==9){
			var_2 = var_2 << var_5;
			currentState = 10;
		}
		else if (currentState ==10){
			var_3 = var_3 | var_2;
			currentState = 11;
		}
		else if (currentState ==11){
			var_4 = var_4 + 1;
			currentState = 12;
		}
		else if (currentState ==12){
			if((var_1!=0)){
				currentState = 13;
			}
			else if((var_1 == 0)){
				currentState = 20;
			}
		}
		else if (currentState ==13){
				var_0=input[0][base_0 + processed_0];
			currentState = 14;
			processed_0++;
		}
		else if (currentState ==14){
			var_2 = var_0 & 127;
			currentState = 15;
		}
		else if (currentState ==15){
			var_1 = var_0 & 128;
			currentState = 16;
		}
		else if (currentState ==16){
			var_5 = var_4 * 7;
			currentState = 17;
		}
		else if (currentState ==17){
			var_2 = var_2 << var_5;
			currentState = 18;
		}
		else if (currentState ==18){
			var_3 = var_3 | var_2;
			currentState = 19;
		}
		else if (currentState ==19){
			var_4 = var_4 + 1;
			currentState = 12;
		}
		else if (currentState ==20){
#ifdef DEBUG
			test_output[printed] = var_3;
			printed++;
#else 
			output[globalID][0] = var_3;
#endif
			currentState = 21;
		}
		else if (currentState ==21){
			var_4 = var_4 & 0;
			currentState = 22;
		}
		else if (currentState ==22){
			var_3 = var_3 & 0;
			currentState = 5;
		}
		else {
			currentState = 23;
		}
		if ((processed_0 >= length_0))
			done = true;
	}
}
void FSTGPU::gvd_baseline_host(){
	printf("Begin gvd_baseline_host \n ");
	printf(" Number of block: %d \n", number_of_block);
	printf(" Number of thread per block: %d\n", number_of_thread);
	hipEvent_t start_execution, stop_execution;
	hipEventCreate( &start_execution);
	hipEventCreate( &stop_execution);
	hipEventRecord( start_execution, 0);
	gvd_baseline_global<<< number_of_block,number_of_thread>>> ( partition_input_base, partition_input_length, input,  partition_output_base,  simulated_output, printed_output);
	hipEventRecord( stop_execution,0);
	hipEventSynchronize(stop_execution);
	float execution_time_ms = 0;
	hipEventElapsedTime(&execution_time_ms, start_execution, stop_execution);
	printf("Execution  Finished\n");
	hipError_t err = hipGetLastError();
	if ( err != hipSuccess )
		printf("CUDA Error: %s\n", hipGetErrorString(err));
	fprintf(stderr,"Execution Time %f ms\n", execution_time_ms);
}
