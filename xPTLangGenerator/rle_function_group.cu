#include "hip/hip_runtime.h"
#include "gpu_fst_engine.h"
#include "shared.h"
typedef struct Return_message{
	uint32_t var_0;
	uint32_t var_1;
	uint32_t nextState;
} Return_message;

__global__
void rle_function_group_global( uint32_t ** input_base, uint32_t ** input_length, uint32_t ** input,
						uint32_t **output_base, uint32_t ** output, uint32_t * test_output){
	uint32_t globalID = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t startStack_shared = threadIdx.x * 0;
	uint32_t base_0 = input_base[globalID][0];
	uint32_t length_0 = input_length[globalID][0];
	uint32_t processed_0 = 0;
	uint32_t currentState = 0;
	uint32_t var_0 =  0;
	uint32_t var_1 =  0;
	uint32_t var_2 =  0;
	uint32_t printed = 0;
	bool done = false;
	uint32_t cycle =0;
	Return_message message;
#ifdef DEBUG
	while ( cycle < 20){
		printf("=============%d=============\n", cycle);
		printf("Current State : %d\n", currentState);
		printf("Var[0]: %d\n", var_0);
		printf("Var[1]: %d\n", var_1);
		printf("Var[2]: %d\n", var_2);
		printf("Processed Input 0: %d/%d\n", processed_0, length_0);
		printf("Current Input: ");
		for ( uint32_t i =0 ; i < 20; i++){
			printf("%d ", input[0][base_0+i]);
}
		printf("\n");
		printf("Current Output: ");
		for ( uint32_t i =0 ; i < printed; i++){
			printf("%d ", test_output[i]);
		}
		printf("\n");
		cycle++;
#else
	while (  (!done) && ( currentState != 5)){
#endif
		if(currentState == 0){
				var_0=input[0][base_0 + processed_0];
			processed_0++;
#ifdef DEBUG
			test_output[printed] = var_0;
			printed++;
#else 
			output[globalID][0] = var_0;
#endif
			var_1 = var_1 + 1;
			currentState = 1;
		}
		else if (currentState ==1){
				var_2=input[0][base_0 + processed_0];
			currentState = 2;
			processed_0++;
		}
		else if (currentState ==2){
			if((var_2 == var_0)){
				currentState = 3;
			}
			else if((var_2!= var_0)){
				currentState = 4;
			}
		}
		else if (currentState ==3){
			var_1 = var_1 + 1;
			currentState = 1;
		}
		else if (currentState ==4){
#ifdef DEBUG
			test_output[printed] = var_1;
			printed++;
#else 
			output[globalID][0] = var_1;
#endif
#ifdef DEBUG
			test_output[printed] = var_2;
			printed++;
#else 
			output[globalID][0] = var_2;
#endif
			var_0 = var_2 + 0;
			var_1 = var_1 & 0;
			var_1 = var_1 + 1;
			currentState = 1;
		}
		else {
			currentState = 5;
		}
		if ((processed_0 >= length_0))
			done = true;
	}
}
void FSTGPU::rle_function_group_host(){
	printf("Begin rle_function_group_host \n ");
	printf(" Number of block: %d \n", number_of_block);
	printf(" Number of thread per block: %d\n", number_of_thread);
	hipEvent_t start_execution, stop_execution;
	hipEventCreate( &start_execution);
	hipEventCreate( &stop_execution);
	hipEventRecord( start_execution, 0);
	rle_function_group_global<<< number_of_block,number_of_thread>>> ( partition_input_base, partition_input_length, input,  partition_output_base,  simulated_output, printed_output);
	hipEventRecord( stop_execution,0);
	hipEventSynchronize(stop_execution);
	float execution_time_ms = 0;
	hipEventElapsedTime(&execution_time_ms, start_execution, stop_execution);
	printf("Execution  Finished\n");
	hipError_t err = hipGetLastError();
	if ( err != hipSuccess )
		printf("CUDA Error: %s\n", hipGetErrorString(err));
	fprintf(stderr,"Execution Time %f ms\n", execution_time_ms);
}
